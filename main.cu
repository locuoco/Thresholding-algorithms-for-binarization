//  A code for binarization using local thresholding algorithms
//  Copyright (C) 2021 Alessandro Lo Cuoco (alessandro.locuoco@gmail.com)

//  This program is free software: you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation, either version 3 of the License, or
//  (at your option) any later version.

//  This program is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.

//  You should have received a copy of the GNU General Public License
//  along with this program.  If not, see <http://www.gnu.org/licenses/>.

/**

	Compilation:
		nvcc main.cu -o ltbin -O3 -std=c++11 -arch=sm_<xy> -I <includes>
	<xy> is the compute capability of the GPU (usually given in the form x.y),
	for example sm_21 corresponds to a compute capability of 2.1.
	<includes> are header files to be included

	example:
		nvcc main.cu -o ltbin -O3 -arch=sm_75

	Compilation with GCC (without CUDA code, extension must be changed to .cpp):
		g++ -std=c++11 -O3 -Wall main.cpp -o ltbincpu

*/

#include <iostream>
#include <string>
#include <chrono>
#include <type_traits>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"

#ifdef __HIPCC__

#define ENABLE_CUDA_GPGPU	// comment this line to use C++ code when compiling with CUDA compiler
							// not needed if you are using GCC
#endif

#include "binarization.hpp"

#ifdef ENABLE_CUDA_GPGPU

#include "binarization_gpu.cuh"

#endif

/* available functions for thresholding:
	== CPU ==
	sauvola
	niblack
	bernsen
	singh
	sauvola_chan
	niblack_chan
	singh_chan
	global

	== GPU ==
	sauvola_gpu
	sauvola_gpu2
	niblack_gpu
	niblack_gpu2
	bernsen_gpu
	bernsen_gpu2
	singh_gpu
	singh_gpu2
	singh_gpu3
	global_gpu
*/

#define FUNC_TO_TEST bernsen
#define FUNC_ARGUMENTS , 15, 100

#define IS_GLOBAL 0 // if you want to test global or global_gpu, please set this to 1
#define IS_SINGH_GPU3 0 // if you want to test singh_gpu3, please set this to 1

#define TO_STR(...) #__VA_ARGS__
#define FUNC_STR(...) TO_STR(__VA_ARGS__)
#define FUNC_NAME FUNC_STR(FUNC_TO_TEST FUNC_ARGUMENTS)

void save_single_image(unsigned char *raw, unsigned int width, unsigned int height, int i, std::string name = std::string("image"))
{
	name += '_';
	name += std::to_string(i);
	name += ".png";
	stbi_write_png(name.c_str(), width, height, 1, raw, width);
}

template <int R = 6>
void simple_test(unsigned char *raw, unsigned int w, unsigned int h)
{
#if defined(ENABLE_CUDA_GPGPU) && IS_GLOBAL == 0 && IS_SINGH_GPU3 == 0
	imgproc::FUNC_TO_TEST<R>(raw, w, h FUNC_ARGUMENTS);
#elif IS_SINGH_GPU3 == 0
	imgproc::FUNC_TO_TEST(raw, w, h, R FUNC_ARGUMENTS);
#else
	imgproc::FUNC_TO_TEST(raw, w, h FUNC_ARGUMENTS);
#endif
}

template <int R = 6>
void prealloc_test(unsigned char *raw, unsigned int w, unsigned int h, int idx)
{
	namespace chrono = std::chrono;

	auto begin = chrono::steady_clock::now();

	simple_test<R>(raw, w, h);

	auto end = chrono::steady_clock::now();

	std::cout << "Time elapsed for processing image " << idx << ": "
			  << chrono::duration_cast<chrono::microseconds>(end - begin).count() * 1.e-6
			  << " [s]\n";

	save_single_image(raw, w, h, idx, FUNC_NAME);

	begin = chrono::steady_clock::now();

	simple_test<R>(raw, w, h);

	end = chrono::steady_clock::now();

	std::cout << "Time elapsed for processing image " << idx << " with preallocation: "
			  << chrono::duration_cast<chrono::microseconds>(end - begin).count() * 1.e-6
			  << " [s]\n";
}

template <int R = 6>
void prealloc_test2(unsigned char *raw, unsigned int w, unsigned int h, int idx)
{
	namespace chrono = std::chrono;

	simple_test<R>(raw, w, h);

	auto begin = chrono::steady_clock::now();

	simple_test<R>(raw, w, h);

	auto end = chrono::steady_clock::now();

	std::cout << chrono::duration_cast<chrono::microseconds>(end - begin).count() * 1.e-6 << ' ';
}

template <int R, int Step, int maxR, std::enable_if<(R < 0)>::type* = nullptr>
void window_test(unsigned char*, unsigned int, unsigned int, int)
{
	std::cout << '\n' << FUNC_NAME << ' ';
}

template <int R = 20, int Step = 2, int maxR = R, std::enable_if<(R >= 0)>::type* = nullptr>
void window_test(unsigned char *raw, unsigned int w, unsigned int h, int idx)
{
	std::cout << '\t' << (maxR % Step + maxR - R)*2+1; // :)
	window_test<R-Step, Step, maxR>(raw, w, h, idx);
	prealloc_test2<R>(raw, w, h, idx);
}

int main(const int narg, const char** args)
{
	namespace chrono = std::chrono;

	auto begin = chrono::steady_clock::now();

#ifdef ENABLE_CUDA_GPGPU
	gpuErrchk(hipFree(nullptr)); // force to create the CUDA context
	std::cout << "CUDA enabled!\n";
#else
	std::cout << "CUDA NOT enabled!\n";
#endif

	auto end = chrono::steady_clock::now();

	std::cout << "Time elapsed for creating CUDA context: "
				  << chrono::duration_cast<chrono::microseconds>(end - begin).count() * 1.e-6
				  << " [s]\n";

	std::string *img_paths;
	unsigned int Nimages;
	if (narg < 2)
	{
		std::cout << "No input image. Nothing to be done. Add the paths/names of the images as arguments to this program." << std::endl;
		return 0;
	}
	else
	{
		Nimages = narg-1;
		img_paths = new std::string[Nimages];
		for (int i = 1; i < narg; ++i)
			img_paths[i-1] = std::string(args[i]);
	}

	for (unsigned int i = 0; i < Nimages; ++i)
	{
		int width, height, ch;

		begin = chrono::steady_clock::now();

		unsigned char *raw = stbi_load(img_paths[i].c_str(), &width, &height, &ch, 1);

		if (!raw)
		{
			std::cerr << "Error: Cannot find " << img_paths[i] << '.' << std::endl;
			continue;
		}

		end = chrono::steady_clock::now();

		std::cout << "Time elapsed for loading image " << i << ": "
				  << chrono::duration_cast<chrono::microseconds>(end - begin).count() * 1.e-6
				  << " [s]\n";

		window_test(raw, width, height, i);

		delete[] raw;
	}

	return 0;
}



























